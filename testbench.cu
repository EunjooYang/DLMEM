/*
   * Writer : Eunjoo Yang
   *
   * Testing the effect of memory layout in GPU
   * cuDNN vs cuda-convnet
*/

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hipDNN.h>  // cudnn
#include <hiprand.h> // hiprand

#define NCHW 0
#define CHWN 1
#define TRUE 1
#define FALSE 0
#define MODE NCHW 
#define DEBUG TRUE
#define CHECK(name,func) printf("\t%s: %s\n",name,hipdnnGetErrorString(func))
#define outputDim(inputDim,pad,filterDim,convolutionStride) 1+(inputDim + 2*pad - filterDim)/convolutionStride
#define FOWARDALGO HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM


int main(){

    
// IF CUDNN MODE
#if MODE==NCHW


    hiprandGenerator_t gen;  //hiprand generator
    hipdnnHandle_t cudnnHandler;
    hipdnnTensorDescriptor_t inputDesc;
    hipdnnTensorDescriptor_t outputDesc;
    hipdnnFilterDescriptor_t filterDesc;
    hipdnnConvolutionDescriptor_t convDesc;

    float *d_input, *h_input;
    float *d_output, *h_output;
    float *d_w; 

    int input_n = 10;
    int input_c = 3;
    int input_h = 28;
    int input_w = 28;
    int filter_cout = 10;
    int filter_cin = input_c;
    int filter_height = 3;
    int filter_width = 3;
    int conv_pad[2] = {0, 0};
    int conv_stride[2] = {1,1};
    int out_w = outputDim(input_w,conv_pad[0],filter_width,conv_stride[0]);
    int out_h = outputDim(input_h,conv_pad[1],filter_height,conv_stride[1]);
    size_t wsSize;
    unsigned int *d_workspace;
    float alpha = 1;
    float beta = 0;


    // Allocate host memory
    h_input = (float*)malloc(input_n*input_c*input_h*input_w*sizeof(float));
    h_output = (float*)malloc(out_w*out_h*filter_cout*input_n*sizeof(float));

    // Allocate device memory
    hipMalloc((void**) &d_input, input_n*input_c*input_h*input_w*sizeof(float));
    hipMalloc((void**) &d_output, out_w*out_h*filter_cout*input_n*sizeof(float));
    hipMalloc((void**) &d_w, filter_cout*filter_height*filter_width*sizeof(float));


// DEBUG MODE : PRINT STATE MESSAGE
#if DEBUG==TRUE

    // Initialize with Random Value in device memory
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
    hiprandGenerate(gen,(unsigned int*)d_input,input_n*input_c*input_h*input_w);
    hiprandGenerate(gen,(unsigned int*)d_w,filter_cout*filter_height*filter_width);

    // initialize the cuDNN library and creates a handle to an opaque structure
    // holding the cuDNN library context
    // cuDNN library context is tied to the current CUDA device
    CHECK("hipdnnCreate",hipdnnCreate(&cudnnHandler));
    // Initialize Generic Tensor Descriptor first
    CHECK("CreateTensorDescriptor", hipdnnCreateTensorDescriptor(&inputDesc));
    // Create TensorDescriptor for 4D Descriptor
    CHECK("TensorDescriptor",hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, input_n, input_c,input_h, input_w));
    // Initialize Generic Tensor Descriptor first
    CHECK("CreateTensorDescriptor", hipdnnCreateTensorDescriptor(&outputDesc));
    // Create TensorDescriptor for 4D Descriptor
    CHECK("TensorDescriptor",hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, input_n, filter_cout,out_h, out_w));
    // Create Filter Descriptor
    CHECK("CreateFilterDescriptor", hipdnnCreateFilterDescriptor(&filterDesc));
    // SetFilter4dDescriptor
    CHECK("SetFilterDescriptor4D",hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, filter_cout, filter_cin, filter_height, filter_width));
    // Create Convolution Descriptor
    CHECK("CreateconvolutionDescriptor",hipdnnCreateConvolutionDescriptor(&convDesc));
    // Set Convolution 2d Descriptor
    CHECK("SetConvolutionDescriptor2D",hipdnnSetConvolution2dDescriptor(convDesc,conv_pad[0],conv_pad[1],conv_stride[0],conv_stride[1],1,1,HIPDNN_CONVOLUTION));

    // Get Convolution Forward Workspace Size
    CHECK("Get Workspace Size",hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandler,inputDesc,filterDesc,convDesc,outputDesc,FOWARDALGO,&wsSize));
    // Generate Workspace Size
    hipMalloc((void**) &d_workspace, wsSize);
    // Convolution Forward
    CHECK("hipdnnConvolutionForward",hipdnnConvolutionForward(cudnnHandler,&alpha,inputDesc,d_input,filterDesc,d_w,convDesc,FOWARDALGO,d_workspace,wsSize,&beta,outputDesc,d_output));
    hipDeviceSynchronize();
    // free host memory
    free(h_input);
    free(h_output);
    // free device memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_w);
    hipFree(d_workspace);

    // Destroy Filter Descriptor
    CHECK("DestroyFilterDescriptor", hipdnnDestroyFilterDescriptor(filterDesc));
    // Destroy Tensor Descriptor
    CHECK("Destroy TensorDescriptor",hipdnnDestroyTensorDescriptor(inputDesc));
    // Destroy cudnnHandler
    CHECK("hipdnnDestroy",hipdnnDestroy(cudnnHandler));
    // Destroy Curand Generator
    hiprandDestroyGenerator(gen);
#else
#endif
#endif
}
