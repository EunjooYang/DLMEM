/*
   * Writer : Eunjoo Yang
   *
   * Testing the effect of memory layout in GPU
   * cuDNN vs cuda-convnet
*/

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hipDNN.h>

#define NCHW 0
#define CHWN 1
#define TRUE 1
#define FALSE 0
#define MODE NCHW 
#define DEBUG TRUE
#define CHECK(name,func) printf("\t%s: %s\n",name,hipdnnGetErrorString(func))


int main(){

    
// IF CUDNN MODE
#if MODE==NCHW

    // Test Parameter
    hipdnnHandle_t cudnnHandler;
    hipdnnTensorDescriptor_t tensorDesc;
    double *d_ts, *h_ts;
    int input_n = 10;
    int input_c = 3;
    int input_h = 28;
    int input_w = 28;

    // Initialize host and device memory
    h_ts = (doube *)malloc(n*c*h*w*sizeof(double));
    hipMalloc((void **)&d_ts, n*c*h*w*sizeof(double));

#if DEBUG==TRUE

    // initialize the cuDNN library and creates a handle to an opaque structure
    // holding the cuDNN library context
    // cuDNN library context is tied to the current CUDA device
    CHECK("hipdnnCreate",hipdnnCreate(&cudnnHandler));
    // Initialize Generic Tensor Descriptor first
    CHECK("CreateTensorDescriptor", hipdnnCreateTensorDescriptor(&tensorDesc));
    // Create TensorDescriptor for 4D Descriptor
    CHECK("TensorDescriptor",hipdnnSetTensor4dDescriptor(tensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, input_n, input_c,input_h, input_w));


    // free host memory
    free(h_ts);
    // free device memory
    hipFree(d_ts);
    // Destroy Tensor Descriptor
    CHECK("Destroy TensorDescriptor",hipdnnDestroyTensorDescriptor(tensorDesc));
    // Destroy cudnnHandler
    CHECK("hipdnnDestroy",hipdnnDestroy(cudnnHandler));
#else


#endif
#endif
}
