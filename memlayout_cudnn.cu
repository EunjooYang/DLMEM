/*
   * memlayout_cudnn.cu
   * 
   * Writer : Eunjoo Yang
   *
   * Testing the effect of memory layout in GPU
   * This code only test the cudnn (NCHW)
   * 
   * How to execute ./memlayout_cudnn minibatchsize input_c input_size output_c filter_size
   *
*/

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hipDNN.h>  // cudnn
#include <hiprand.h> // hiprand
#include <sys/time.h> // get time of day

#define NCHW 0
#define CHWN 1
#define TRUE 1
#define FALSE 0
#define MODE NCHW 
#define DEBUG FALSE
// MACRO for debugging
#define CHECK(name,func) printf("\t%s: %s\n",name,hipdnnGetErrorString(func))    
// MACRO to calculate output dimension of filter
#define outputDim(inputDim,pad,filterDim,convolutionStride) 1+(inputDim + 2*pad - filterDim)/convolutionStride
// convolution method
#define FOWARDALGO HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM


int main(int argc, char *argv[]){

    
    hiprandGenerator_t gen;  //hiprand generator
    hipdnnHandle_t cudnnHandler;
    hipdnnTensorDescriptor_t inputDesc;
    hipdnnTensorDescriptor_t outputDesc;
    hipdnnFilterDescriptor_t filterDesc;
    hipdnnConvolutionDescriptor_t convDesc;

    float *d_input, *h_input;
    float *d_output, *h_output;
    float *d_w; 

    int input_n = 256;
    int input_c = 3;
    int input_h = 24;
    int input_w = 24;
    int filter_cout = 64;
    int filter_cin = input_c;
    int filter_height = 5;
    int filter_width = 5;
    int conv_pad[2] = {0, 0};
    int conv_stride[2] = {1,1};
    int out_w = outputDim(input_w,conv_pad[0],filter_width,conv_stride[0]);
    int out_h = outputDim(input_h,conv_pad[1],filter_height,conv_stride[1]);
    size_t wsSize;
    unsigned int *d_workspace;
    float alpha = 1;
    float beta = 0;
    struct timeval start_point, end_point;
    double elapsed_time;

    if (argc < 6){
        printf("Execution Error\n");
        printf("Please Enter the Execution Options \n");
        printf("./memlayout_cudnn 'minibatchsize' 'input_c' 'input_size' 'output_c' 'filter_size'\n");
        printf("Follow Default Setting\n");
        printf("======================================\n");
        printf("Start to cudnn memory layout test\n");
        printf("\t mini batch size : %d\n",input_n);
        printf("\t input channel : %d\n", input_c);
        printf("\t input image size: %d\n", input_w);
        printf("\t output channel : %d\n", filter_cout);
        printf("\t filter size : %d\n", filter_height);
        printf("======================================\n");
    }else{

        input_n = atoi(argv[1]);  
        input_c = atoi(argv[2]);
        input_w = input_h =  atoi(argv[3]);
        filter_cout = atoi(argv[4]);
        filter_height = filter_width = atoi(argv[5]);
        printf("======================================\n");
        printf("Start to cudnn memory layout test\n");
        printf("\t mini batch size : %d\n",input_n);
        printf("\t input channel : %d\n", input_c);
        printf("\t input image size: %d\n", input_w);
        printf("\t output channel : %d\n", filter_cout);
        printf("\t filter size : %d\n", filter_height);
        printf("======================================\n");
    }

    // Allocate host memory
    h_input = (float*)malloc(input_n*input_c*input_h*input_w*sizeof(float));
    h_output = (float*)malloc(out_w*out_h*filter_cout*input_n*sizeof(float));

    // Allocate device memory
    hipMalloc((void**) &d_input, input_n*input_c*input_h*input_w*sizeof(float));
    hipMalloc((void**) &d_output, out_w*out_h*filter_cout*input_n*sizeof(float));
    hipMalloc((void**) &d_w, filter_cout*filter_height*filter_width*sizeof(float));


// DEBUG MODE : PRINT STATE MESSAGE
#if DEBUG==TRUE

    // Initialize with Random Value in device memory
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
    hiprandGenerate(gen,(unsigned int*)d_input,input_n*input_c*input_h*input_w);
    hiprandGenerate(gen,(unsigned int*)d_w,filter_cout*filter_height*filter_width);

    // initialize the cuDNN library and creates a handle to an opaque structure
    // holding the cuDNN library context
    // cuDNN library context is tied to the current CUDA device
    CHECK("hipdnnCreate",hipdnnCreate(&cudnnHandler));
    // Initialize Generic Tensor Descriptor first
    CHECK("CreateTensorDescriptor", hipdnnCreateTensorDescriptor(&inputDesc));
    // Create TensorDescriptor for 4D Descriptor
    CHECK("TensorDescriptor",hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, input_n, input_c,input_h, input_w));
    // Initialize Generic Tensor Descriptor first
    CHECK("CreateTensorDescriptor", hipdnnCreateTensorDescriptor(&outputDesc));
    // Create TensorDescriptor for 4D Descriptor
    CHECK("TensorDescriptor",hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, input_n, filter_cout,out_h, out_w));
    // Create Filter Descriptor
    CHECK("CreateFilterDescriptor", hipdnnCreateFilterDescriptor(&filterDesc));
    // SetFilter4dDescriptor
    CHECK("SetFilterDescriptor4D",hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, filter_cout, filter_cin, filter_height, filter_width));
    // Create Convolution Descriptor
    CHECK("CreateconvolutionDescriptor",hipdnnCreateConvolutionDescriptor(&convDesc));
    // Set Convolution 2d Descriptor
    CHECK("SetConvolutionDescriptor2D",hipdnnSetConvolution2dDescriptor(convDesc,conv_pad[0],conv_pad[1],conv_stride[0],conv_stride[1],1,1,HIPDNN_CONVOLUTION));

    // Get Convolution Forward Workspace Size
    CHECK("Get Workspace Size",hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandler,inputDesc,filterDesc,convDesc,outputDesc,FOWARDALGO,&wsSize));
    // Generate Workspace Size
    hipMalloc((void**) &d_workspace, wsSize);

    // measure the time
    gettimeofday(&start_point,NULL);
    // Convolution Forward
    CHECK("hipdnnConvolutionForward",hipdnnConvolutionForward(cudnnHandler,&alpha,inputDesc,d_input,filterDesc,d_w,convDesc,FOWARDALGO,d_workspace,wsSize,&beta,outputDesc,d_output));
    // wait until done
    hipDeviceSynchronize(); 
    // measure the finish time
    gettimeofday(&end_point,NULL);
    elapsed_time = (double)(end_point.tv_sec)*1000+(double)(end_point.tv_usec)/1000-(double)(start_point.tv_sec)*1000-(double)(start_point.tv_usec)/1000;
    printf("Elapsed Time : %f ms\n",elapsed_time);


    // free host memory
    free(h_input);
    free(h_output);
    // free device memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_w);
    hipFree(d_workspace);

    // Destroy Filter Descriptor
    CHECK("DestroyFilterDescriptor", hipdnnDestroyFilterDescriptor(filterDesc));
    // Destroy Tensor Descriptor
    CHECK("Destroy TensorDescriptor",hipdnnDestroyTensorDescriptor(inputDesc));
    // Destroy cudnnHandler
    CHECK("hipdnnDestroy",hipdnnDestroy(cudnnHandler));
    // Destroy Curand Generator
    hiprandDestroyGenerator(gen);
#else

    // Initialize with Random Value in device memory
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
    hiprandGenerate(gen,(unsigned int*)d_input,input_n*input_c*input_h*input_w);
    hiprandGenerate(gen,(unsigned int*)d_w,filter_cout*filter_height*filter_width);

    // initialize the cuDNN library and creates a handle to an opaque structure
    // holding the cuDNN library context
    // cuDNN library context is tied to the current CUDA device
    hipdnnCreate(&cudnnHandler);
    // Initialize Generic Tensor Descriptor first
    hipdnnCreateTensorDescriptor(&inputDesc);
    // Create TensorDescriptor for 4D Descriptor
    hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, input_n, input_c,input_h, input_w);
    // Initialize Generic Tensor Descriptor first
    hipdnnCreateTensorDescriptor(&outputDesc);
    // Create TensorDescriptor for 4D Descriptor
    hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, input_n, filter_cout,out_h, out_w);
    // Create Filter Descriptor
    hipdnnCreateFilterDescriptor(&filterDesc);
    // SetFilter4dDescriptor
    hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, filter_cout, filter_cin, filter_height, filter_width);
    // Create Convolution Descriptor
    hipdnnCreateConvolutionDescriptor(&convDesc);
    // Set Convolution 2d Descriptor
    hipdnnSetConvolution2dDescriptor(convDesc,conv_pad[0],conv_pad[1],conv_stride[0],conv_stride[1],1,1,HIPDNN_CONVOLUTION);

    // Get Convolution Forward Workspace Size
    hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandler,inputDesc,filterDesc,convDesc,outputDesc,FOWARDALGO,&wsSize);
    // Generate Workspace Size
    hipMalloc((void**) &d_workspace, wsSize);

    // measure the time
    gettimeofday(&start_point,NULL);
    // Convolution Forward
    hipdnnConvolutionForward(cudnnHandler,&alpha,inputDesc,d_input,filterDesc,d_w,convDesc,FOWARDALGO,d_workspace,wsSize,&beta,outputDesc,d_output);
    // wait until done
    hipDeviceSynchronize(); 
    // measure the finish time
    gettimeofday(&end_point,NULL);
    elapsed_time = (double)(end_point.tv_sec)*1000+(double)(end_point.tv_usec)/1000-(double)(start_point.tv_sec)*1000-(double)(start_point.tv_usec)/1000;
    printf(" Elapsed Time : %f ms\n",elapsed_time);


    // free host memory
    free(h_input);
    free(h_output);
    // free device memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_w);
    hipFree(d_workspace);

    // Destroy Filter Descriptor
    hipdnnDestroyFilterDescriptor(filterDesc);
    // Destroy Tensor Descriptor
    hipdnnDestroyTensorDescriptor(inputDesc);
    // Destroy cudnnHandler
    hipdnnDestroy(cudnnHandler);
    // Destroy Curand Generator
    hiprandDestroyGenerator(gen);
#endif
}
